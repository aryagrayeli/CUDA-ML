#pragma once
#include <hip/hip_runtime.h>


#include <stdlib.h>
#include <math.h>

// Spawn N Threads
__global__ void matrix_mul(float * a, float * b, float * output, int N, int M) {
    // int n = gridDim.x;
    // int m = blockDim.x;
    // int r = blockIdx.x;
    // int c = threadIdx.x;

    // int Mi = 31 - __builtin_clz(m);
    // int M = 1 << Mi;
    // __shared__ float arr[M];
    // __shared__ float B[m];
    // cudaMemcpy(arr, a[r], sizeof(float) * m, cudaMemcpyDeviceToDevice);
    // cudaMemcpy(B, b, sizeof(float) * m, cudaMemcpyDeviceToDevice);
	// cudaMemset(arr + m, 0, sizeof(float) * (M - m));
    // arr[c] *= B[c];
    // __syncthreads();
    // for (int i = 0; i < Mi; i++) {
    //     if (c >> (Mi - i - 1)) {
    //         continue;
    //     }
    //     int a = c << (i + 1);
    //     int b = a + (1 << i);
    //     arr[a] += arr[b];
    //     __syncthreads();
    // }

    // output[r] = *arr; 
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < N) {
        float sum = 0;
        for(int i = 0; i < M; i++) 
            sum += a[row * M + i] * b[i];
        output[row] = sum;
    }
}

// Spawn N Threads
__global__ void vector_sigmoid(float * input, float * output, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N)
        output[i] = 1.0 / (1 + exp(-input[i]));
}

// Spawn N Threads
__global__ void vector_dsigmoid(float * input, float * output, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N) {
        float sigmoid = 1.0 / (1 + exp(-input[i]));
        output[i] = sigmoid * (1-sigmoid);
    }
}

// Spawn N Threads
__global__ void vector_relu(float * input, float * output, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N)
        output[i] = input[i] > 0 ? input[i] : 0;
}

// Spawn N Threads
__global__ void vector_drelu(float * input, float * output, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N)
        output[i] = input[i] > 0;
}

// Spawn N Threads
__global__ void vector_tanh(float * input, float * output, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N)
        output[i] = (exp(input[i]) - exp(-input[i])) / (exp(input[i]) + exp(-input[i]));
}

// Spawn N Threads
__global__ void vector_dtanh(float * input, float * output, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N) {
        float tanh = (exp(input[i]) - exp(-input[i])) / (exp(input[i]) + exp(-input[i]));
        output[i] = 1 - tanh*tanh;
    }
}

float* vector_softmax(float* input, int n) {
  float* output = (float*) malloc(sizeof(float) * n);

  float sum = 0;
  for(int i = 0; i < n; i++)
    sum += exp(input[i]);
  for(int i = 0; i < n; i++)
    output[i] = exp(input[i])/sum;
  
  return output;
}

float* vector_dsoftmax(float* input, int j, int n) {
  float* output = (float*) malloc(sizeof(float) * n);

  float sum = 0;
  for(int i = 0; i < n; i++)
    sum += exp(input[i]);
  float sj = exp(input[j])/sum;
  for(int i = 0; i < n; i++)
    if(i == j)
      output[i] = sj * (1-sj);
    else
      output[i] = -sj*(exp(input[i])/sum);
  
  return output;
}

// Spawn N Threads
__global__ void vector_add(float * a, float * b, float * output, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N)
        output[i] = a[i] + b[i];
}

// Spawn N Threads
__global__ void matrix_hadamard(float * a, float * b, float * output, int N, int M) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row < N && col < M) 
        output[row * M + col] = a[row * M + col] * b[row * M + col];
}

// Spawn N x M Threads
__global__ void matrix_trans(float * input, float * output, int N, int M) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row < N && col < M) 
        output[col * N + row] = input[row * M + col];
}

// Spawn N x M Threads
__global__ void matrix_scalar(float * input, int sc, float * output, int N, int M) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row < N && col < M) 
        output[row * M + col] = sc * input[row * M + col];
}