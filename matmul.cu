#include "hip/hip_runtime.h"
#pragma once

#include <stdlib.h>
#include <math.h>

// Spawn N x M Threads
__global__ void matrix_mul(float ** a, float * b, float * output) {
    int n = gridDim.x;
    int m = blockDim.x;
    int r = blockIdx.x;
    int c = threadIdx.x;

    int Mi = 31 - __builtin_clz(m);
    int M = 1 << Mi;
    __shared__ float arr[M];
    __shared__ float B[m];
    hipMemcpy(arr, a[r], sizeof(float) * m, hipMemcpyDeviceToDevice);
    hipMemcpy(B, b, sizeof(float) * m, hipMemcpyDeviceToDevice);
	hipMemset(arr + m, 0, sizeof(float) * (M - m));
    arr[c] *= B[c];
    __syncthreads();
    for (int i = 0; i < Mi; i++) {
        if (c >> (Mi - i - 1)) {
            continue;
        }
        int a = c << (i + 1);
        int b = a + (1 << i);
        arr[a] += arr[b]
        __syncthreads();
    }

    output[r] = *arr; 
}

// Spawn 1 x N Threads
__global__ void vector_sigmoid(float * input, float * output) {
    int i = threadIdx.x;
    output[i] = 1.0 / (1 + exp(-input[i]));
}

// Spawn 1 x N Threads
__global__ void vector_dsigmoid(float * input, float * output) {
    int i = threadIdx.x;
    float sigmoid = 1.0 / (1 + exp(-input[i]));
    output[i] = sigmoid * (1-sigmoid);
}

// Spawn 1 x N Threads
__global__ void vector_relu(float * input, float * output) {
    int i = threadIdx.x;
    output[i] = input[i] > 0 ? input[i] : 0;
}

// Spawn 1 x N Threads
__global__ void vector_drelu(float * input, float * output) {
    int i = threadIdx.x;
    output[i] = input[i] > 0;
}

// Spawn 1 x N Threads
__global__ void vector_tanh(float * input, float * output) {
    int i = threadIdx.x;
    output[i] = (exp(input[i]) - exp(-input[i])) / (exp(input[i]) + exp(-input[i]));
}

// Spawn 1 x N Threads
__global__ void vector_dtanh(float * input, float * output) {
    int i = threadIdx.x;
    float tanh = (exp(input[i]) - exp(-input[i])) / (exp(input[i]) + exp(-input[i]));
    output[i] = 1 - tanh*tanh;
}

float* vector_softmax(float* input, int n) {
  float* output = (float*) malloc(sizeof(float) * n);

  float sum = 0;
  for(int i = 0; i < n; i++)
    sum += exp(input[i]);
  for(int i = 0; i < n; i++)
    output[i] = exp(input[i])/sum;
  
  return output;
}

float* vector_dsoftmax(float* input, int j, int n) {
  float* output = (float*) malloc(sizeof(float) * n);

  float sum = 0;
  for(int i = 0; i < n; i++)
    sum += exp(input[i]);
  float sj = exp(input[j])/sum;
  for(int i = 0; i < n; i++)
    if(i == j)
      output[i] = sj * (1-sj);
    else
      output[i] = -sj*(exp(input[i])/sum);
  
  return output;
}

// Spawn 1 x N Threads
__global__ void vector_add(float * a, float * b, float * output) {
    int i = threadIdx.x;
    output[i] = a[i] + b[i];
}

// Spawn N x M Threads
__global__ void matrix_hadamard(float ** a, float ** b, float ** output) {
    int r = blockIdx.x;
    int c = threadIdx.x;
    output[r][c] = a[r][c] * b[r][c];
}

// Spawn N x M Threads
__global__ void matrix_trans(float ** input, float ** output) {
    int r = threadIdx.x;
    int c = blockIdx.x;
    output[r][c] = input[c][r];
}

// Spawn N x M Threads
__global__ void matrix_scalar(float ** input, int sc, float ** output) {
    int r = blockIdx.x;
    int c = threadIdx.x;
    output[r][c] = sc * input[r][c];
}