#pragma once

#include <stdlib.h>
#include <math.h>

int main() {
    int n = 3, m = 3;

    float** A = (float**) malloc(sizeof(float*) * n);
    for(int i = 0; i < n; i++) A[i] = (float*) malloc(sizeof(float) * m);

    // [1 -2 3]
    // [4 5 6]
    // [-7 8 9]

    A[0][0] = 1; A[0][1] = -2; A[0][2] = 3; A[1][0] = 4; A[1][1] = 5; A[1][2] = 6; A[2][0] = -7; A[2][1] = 8; A[2][2] = 9;

    float** A_cuda;
    hipMalloc(&A_cuda, sizeof(float*) * n);
    for (int i = 0; i < n; i++) {
        hipMalloc(&A_cuda[i], sizeof(float) * m);
        hipMemcpy(A_cuda[i], A[i], sizeof(float) * m, hipMemcpyHostToDevice);
    }

    float* B = (float*) malloc(sizeof(float) * n);

    // [-1 2 3]

    B[0] = -1; B[1] = 2; B[2] = 3;

    float* B_cuda;
    hipMalloc(&B_cuda, sizeof(float) * n);
    hipMemcpy(B_cuda, B, sizeof(float) * n, hipMemcpyHostToDevice);

    float* output_cuda;
    hipMalloc(&output_cuda, sizeof(float) * n);

    matrix_mul<<<n, m>>>(A_cuda, B_cuda, output_cuda);

    float* output;
    hipMemcpy(output, output_cuda, sizeof(float) * n, hipMemcpyDeviceToHost);

    printf("Test Matrix Multiplication\n");
    printf("Output: %f, %f, %f; Should be: 4, 24, 50\n\n", output[0], output[1], output[2]);
}