#pragma once

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "matmul.cu"
#include "matmul.h"
#include <time.h>

int main() {

  srand(time(NULL));
  
  int power = 15;
  int bigN = (1<<power), bigM = (1<<14); 
  float *bigA, *bigB, *bigOutput;
  hipMallocManaged(&bigA, bigN * bigM * sizeof(float));
  hipMallocManaged(&bigB, bigM * sizeof(float));
  hipMallocManaged(&bigOutput, bigN * sizeof(float));

  printf("successfully malloced a, b, output");

  for(int i=0;i<bigN*bigM;i++) bigA[i] = rand()/RAND_MAX;
  for(int i=0;i<bigM;i++) bigB[i] = rand()/RAND_MAx;
  
  printf("Test Big Matrix Multiplication\n");
  int threads = 1 << 6;
  dim3 bigGridSize((bigN + threads - 1)/threads, (bigM + threads - 1)/threads, 1);
  dim3 bigBlockSize(threads, threads, 1);
  matrix_mul<<<bigGridSize, bigBlockSize>>>(bigA, bigB, bigOutput, bigN, bigM);
  hipDeviceSynchronize();
  printf("Done executing GPU matmul\n");

  for(int i=0;i<bigN;i++) printf("%f ",bigOutput[i]);
  printf("\n");
  
}	
